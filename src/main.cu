#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <memory>

#include <iomanip>
#include <ctime>
#include <sstream>

#include <iostream>
#include <pthread.h>

#include "glad/glad.h"
#include <GLFW/glfw3.h>

#include <chrono>

#include "OctreeSerializer.cuh"
#include "render.cuh"

#include "cuda_common/hip/hip_runtime_api.h"

#include <hipcub/hipcub.hpp>
#include <cub/util_allocator.cuh>
#include <cub/device/device_radix_sort.cuh>

#include "CameraLoader.h"

#include <deque>

#include <thrust/count.h>
#include <thrust/device_vector.h>

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"
#include "raster_helper.cuh"
#include "HybridVH.h"


#define N 100000000
#define MAX_ERR 1e-6

int SCREEN_WIDTH = 1920;
int SCREEN_HEIGHT = 1080;

const int FPS_COUNTER_REFRESH = 60;

glm::mat3 cameraRotation;
glm::mat4 modelview;
glm::mat4 perspective;

glm::vec3 cameraPosition = glm::vec3(0.0f);
float angleDirection = 0.0f;
glm::vec3 lookDirection = glm::vec3(1.0f, 0.0f, 0.0f);
glm::vec3 rightDirection = glm::vec3(0.0f, 0.0f, 1.0f);
const float movement_step = 0.1f;

static void error_callback(int error, const char* description)
{
    fprintf(stderr, "Error: %s\n", description);
}

int markForRender(bool *renderMask, std::vector<HybridVH *> & nodes, int renderLevel, glm::vec3 &cameraPosition, float fovy, int SW, float dpt, int numSplats)
{
	int rendered = 0;
    omp_set_num_threads(6);
    // #pragma omp parallel for
	for(int i = 0; i < nodes.size(); i++){
        HybridVH * tree_root = nodes[i];
        std::deque<HybridVH *> process_queue;
        process_queue.push_back(tree_root);
        while(!process_queue.empty()){
            HybridVH * node = process_queue.front();
            process_queue.pop_front();
            if (renderLevel == -1)
            {
                int shouldRenderNode = 0;
                if (node == nullptr)
                    continue;
                /* Easiest implementation, maximum projection by distance */
                float S = glm::length(node->coverage[0] - node->coverage[1]);
                float D = glm::length((node->coverage[0] + node->coverage[1]) / 2.0f - cameraPosition);

                float P = S / D * (SW / fovy);

                shouldRenderNode = (P > dpt);

                if (shouldRenderNode)
                { // is node big enough on the screen?
                    if (node->containedSplats->size() > 0)
                    {
                        for (uint32_t splat : *(node->containedSplats))
                            renderMask[splat] = true;
                    }
                    else
                    {
                        for(auto child : node ->children){
                            process_queue.push_back(child);
                        }
                    }
                }
                else
                {
                    if (node->representative != 0)
                    {
                        renderMask[node->representative] = true;
                    }
                    else
                    {
                        if(node->children.size() == 0){
                            for(auto splat : *(node->containedSplats)){
                                renderMask[splat] = 1;
                            }
                        }
                        for(auto child : node->children){
                            process_queue.push_back(child);
                        }
                    }
                }
            }
            else
            {
                if (node->level == renderLevel && node->representative != 0)
                {
                    renderMask[node->representative] = true;
                }
                if (node->level < renderLevel && node->isLeaf)
                {
                    for (uint32_t splat : *(node->containedSplats)){
                        renderMask[splat] = true;
                    }
                }
                else if(node->level < renderLevel){
                    for(HybridVH * child : node->children){
                        process_queue.push_back(child);
                    }
                }
            }
        }
	}
    
    for(int i = 0; i < numSplats; i++){
        if(renderMask[i]){
            rendered++;
        }
    }

    return rendered;
}
 
template <typename T>
int markForRender(bool *renderMask, T * node, int renderLevel, glm::vec3 &cameraPosition, float fovy, int SW, float dpt, int numSplats)
{
	int rendered = 0;
        T * tree_root = node;
        std::deque<T *> process_queue;
        process_queue.push_back(tree_root);
        while(!process_queue.empty()){
            T * node = process_queue.front();
            process_queue.pop_front();
            if (renderLevel == -1)
            {
                int shouldRenderNode = 0;
                if (node == nullptr)
                    continue;
                /* Easiest implementation, maximum projection by distance */
                float S = glm::length(node->coverage[0] - node->coverage[1]);
                float D = glm::length((node->coverage[0] + node->coverage[1]) / 2.0f - cameraPosition);

                float P = S / D * (SW / fovy);

                shouldRenderNode = (P > dpt);

                if (shouldRenderNode)
                { // is node big enough on the screen?
                    if (node->containedSplats->size() > 0)
                    {
                        for (uint32_t splat : *(node->containedSplats))
                            renderMask[splat] = true;
                    }
                    else
                    {
                        for(auto child : node ->children){
                            process_queue.push_back(child);
                        }
                    }
                }
                else
                {
                    if (node->representative != 0)
                    {
                        renderMask[node->representative] = true;
                    }
                    else
                    {
                        if(node->children.size() == 0){
                            for(auto splat : *(node->containedSplats)){
                                renderMask[splat] = 1;
                            }
                        }
                        for(auto child : node->children){
                            process_queue.push_back(child);
                        }
                    }
                }
            }
            else
            {
                if (node->level == renderLevel && node->representative != 0)
                {
                    renderMask[node->representative] = true;
                }
                if (node->level < renderLevel && node->isLeaf)
                {
                    for (uint32_t splat : *(node->containedSplats)){
                        renderMask[splat] = true;
                    }
                }
                else if(node->level < renderLevel){
                    for(T * child : node->children){
                        process_queue.push_back(child);
                    }
                }
            }
        }
    
    for(int i = 0; i < numSplats; i++){
        if(renderMask[i]){
            rendered++;
        }
    }

    return rendered;
}

static void key_callback(GLFWwindow* window, int key, int scancode, int action, int mods)
{
    if (key == GLFW_KEY_ESCAPE && action == GLFW_PRESS)
        glfwSetWindowShouldClose(window, GLFW_TRUE);
    if(key == GLFW_KEY_Z && (action == GLFW_REPEAT || action == GLFW_PRESS)){
        cameraPosition += cameraRotation* glm::vec3(0.0f, movement_step, 0.0f);
    }
    if(key == GLFW_KEY_X && (action == GLFW_REPEAT || action == GLFW_PRESS)){
        cameraPosition += cameraRotation* glm::vec3(0.0f, -movement_step, 0.0f);
    }
    if(key == GLFW_KEY_W && (action == GLFW_REPEAT || action == GLFW_PRESS)){
        cameraPosition += movement_step * cameraRotation* rightDirection;
    }
    if(key == GLFW_KEY_S && (action == GLFW_REPEAT || action == GLFW_PRESS)){
        cameraPosition -= movement_step * cameraRotation* rightDirection;
    }
    if(key == GLFW_KEY_D && (action == GLFW_REPEAT || action == GLFW_PRESS)){
        cameraPosition += movement_step * cameraRotation* lookDirection;
    }
    if(key == GLFW_KEY_A && (action == GLFW_REPEAT || action == GLFW_PRESS)){
        cameraPosition -= movement_step * cameraRotation* lookDirection;
    }
    // if(key == GLFW_KEY_Q && (action == GLFW_REPEAT || action == GLFW_PRESS)){
    //     angleDirection -= 0.01f;
    //     lookDirection  = glm::vec3(sin(angleDirection), 0.f, cos(angleDirection));
    //     rightDirection = glm::vec3(-cos(angleDirection), 0.f, sin(angleDirection));
    // }
    // if(key == GLFW_KEY_E && (action == GLFW_REPEAT || action == GLFW_PRESS)){
    //     angleDirection += 0.01f;
    //     lookDirection  = glm::vec3(sin(angleDirection), 0.f, cos(angleDirection));
    //     rightDirection = glm::vec3(-cos(angleDirection), 0.f, sin(angleDirection));
    // }
    if(key == GLFW_KEY_U && (action == GLFW_REPEAT || action == GLFW_PRESS)){
        cameraIndex = std::max(0, cameraIndex - 1);
    }
    if(key == GLFW_KEY_I && (action == GLFW_REPEAT || action == GLFW_PRESS)){
        cameraIndex = std::min(300, cameraIndex + 1);
    }
}

void framebuffer_size_callback(GLFWwindow* window, int width, int height)
{
    glViewport(0, 0, width, height);
}

void initGLContextAndWindow(GLFWwindow** window){
    
    
    glfwSetErrorCallback(error_callback);
 
    if (!glfwInit())
        exit(EXIT_FAILURE);
 
    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 2);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 0);
 
    *window = glfwCreateWindow(SCREEN_WIDTH, SCREEN_HEIGHT, "AccelerateGS", NULL, NULL);
    if (!window)
    {
        glfwTerminate();
        exit(EXIT_FAILURE);
    }
 
    glfwSetKeyCallback(*window, key_callback);
    glfwSetFramebufferSizeCallback(*window, framebuffer_size_callback);
 
    glfwMakeContextCurrent(*window);
    gladLoadGLLoader((GLADloadproc)glfwGetProcAddress);
    glfwSwapInterval(0);

    setupIMGui(window);
}

struct ThreadPayload{
    SpacePartitioningBase * spacePartitioningRoot;
    std::vector<SplatData> * sd;
    int * num_elements;
    volatile int * progress;
};

void * spacePartitioningThread(void * input){
    ThreadPayload * payload = static_cast<ThreadPayload *>(input);
    payload->spacePartitioningRoot->buildVHStructure(*(payload->sd), *(payload->num_elements), payload->progress);
    pthread_exit(NULL);
}

template <typename T>
void splitTree(std::vector<T*> & nodes, SpacePartitioningBase * spbase){
    /*
        Split structure in shallower subtrees.
        This allows independent marking on each subtree on the GPU.
    */
    T * sparse_node = new T();
    sparse_node->isLeaf = true;
    sparse_node->representative = 0;
    sparse_node->levelType = OctreeLevel;

    std::deque<T *> q_nodes;
    q_nodes.push_back((T*)(spbase));

    while(!q_nodes.empty()){
        T * crt_node = q_nodes.front();
        q_nodes.pop_front();
        for(T* child : crt_node->children){
            if(child->levelType == BipartitionLevel){
                nodes.push_back(child);
            }
        }
        for(T* child : crt_node->children){
            if(child->levelType == OctreeLevel){
                if(child->isLeaf){
                    for(auto splat : *(child->containedSplats))
                    sparse_node->containedSplats->push_back(splat);
                }
                else{
                    q_nodes.push_back(child);
                }
            }    
        }
    }
    nodes.push_back(sparse_node);
    printf("Sparse node size: %d\n", sparse_node->containedSplats->size());

}

int main(){
    GLFWwindow* window;

    std::chrono::steady_clock::time_point begin;
    std::chrono::steady_clock::time_point end;

    loadCameraFile("../../models/train/cameras.json");
    loadGenericProperties(SCREEN_WIDTH, SCREEN_HEIGHT, fovx, fovy);

    loadApplicationConfig("../config.cfg", renderConfig);

    numCameraPositions = cameraData.size();

    initGLContextAndWindow(&window);

    /* Load splat scene data from file */
    std::vector<SplatData> sd;
    bool * renderMask;
    int num_elements = 0;
    int res = loadSplatData("../../models/train/point_cloud/iteration_small/point_cloud.ply", sd, &num_elements);
    printf("Loaded %d splats from file\n", num_elements);

    const uint32_t orig_num_splats = num_elements;

    const uint32_t maxDuplicatedGaussians = num_elements * 64;

    // First of all, build da octree
    begin = std::chrono::steady_clock::now();
    #if defined(_OPENMP)
        printf("Using OpenMP, yey\n");
    #endif

    /* OpenGL configuration */
    glPixelStorei(GL_UNPACK_ALIGNMENT, 16);      // 4-byte pixel alignment

    glClearColor(0, 0, 0, 0);                   // background color
    glClearStencil(0);                          // clear stencil buffer
    glClearDepth(1.0f);                         // 0 is near, 1 is far
    glEnable(GL_BLEND);  
    glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);  


    volatile int progress = 0;
    float progressmax = 16.0f;
    SpacePartitioningBase * spacePartitioningRoot = (nullptr);
    if(renderConfig.structure == std::string("octree")){
        spacePartitioningRoot = new GaussianOctree();
        progressmax = 256.0f;
    }
    else if(renderConfig.structure == std::string("bvh")){
        spacePartitioningRoot = new GaussianBVH();
    }
    else if(renderConfig.structure == std::string("hybrid")){
        spacePartitioningRoot = new HybridVH();
        progressmax = 256.0f;
    }

    if(spacePartitioningRoot == nullptr){
        printf("This ain't good lol....\n");
    }

    octreeLevel = renderConfig.octreeLevel;


    /* Compute space partitioning in a separate PThread */
    pthread_t t_id;
    ThreadPayload payload;
    payload.num_elements = &num_elements;
    payload.progress = &progress;
    payload.spacePartitioningRoot = spacePartitioningRoot;
    payload.sd = &sd;

#if true
    pthread_create(&t_id, NULL, spacePartitioningThread, (void *)(&payload));

    while(progress!=1024){
        /* Clear color and depth buffers */
        glClear(GL_DEPTH_BUFFER_BIT | GL_COLOR_BUFFER_BIT);
        buildLoadingInterface(progress / progressmax);
        renderInterface();
        /* Swap buffers and handle GLFW events */
        glfwSwapBuffers(window);
        glfwPollEvents();
    }

    pthread_join(t_id, NULL);
#else
    spacePartitioningRoot->buildVHStructure(sd, num_elements, &progress);
#endif
    // std::vector<HybridVH *> nodes;
    // splitTree<HybridVH>(nodes, spacePartitioningRoot);

    // /* Put node pointers into array for CUDA processing */

    // size_t totalStorageSize = 0;
    // size_t nodeHeaderSize = sizeof(CUDATreeNode);

    // for(int i = 0; i < nodes.size(); i++){
    //     HybridVH * tree_root = nodes[i];
    //     std::deque<HybridVH *> process_queue;
    //     process_queue.push_back(tree_root);
    //     while(!process_queue.empty()){
    //         HybridVH * node = process_queue.front();

    //         totalStorageSize += 1;
            
    //         process_queue.pop_front();
    //         for(HybridVH * child : node->children){
    //             process_queue.push_back(child);
    //         }

    //     }
	// }

    // /* Get the necessary memory to serialize in RAM the array of subtrees */
    // CUDATreeNode * storageBlock = (CUDATreeNode *)malloc(totalStorageSize * sizeof(CUDATreeNode));
    // CUDATreeNode * cudaStorageBlock = NULL;
    // uint32_t * cuda_roots = NULL;

    // size_t currentMemoryPosition = 0;
    // std::vector<uint32_t> roots;

    // for(int i = 0; i < nodes.size(); i++){
    //     HybridVH * tree_root = nodes[i];
    //     roots.push_back(currentMemoryPosition);
    //     std::deque<HybridVH *> process_queue;
    //     process_queue.push_back(tree_root);
    //     while(!process_queue.empty()){
    //         HybridVH * node = process_queue.back();
    //         process_queue.pop_back();

    //         size_t child1Pos = 0, child2Pos = 0;
    //         if(node->children.size() > 0){
    //             child1Pos = process_queue.size() + currentMemoryPosition + 1;
    //         }
    //         if(node->children.size() > 1){
    //             child2Pos = process_queue.size() + currentMemoryPosition + 2;
    //         }

    //         storageBlock[currentMemoryPosition].childrenIndices[0] = child1Pos;
    //         storageBlock[currentMemoryPosition].childrenIndices[1] = child2Pos;

    //         for(int s = 0; s < sizeof(storageBlock[currentMemoryPosition].splatIds) / sizeof(uint32_t); s++){
    //             storageBlock[currentMemoryPosition].splatIds[s] = 0;
    //         }

    //         for(int s = 0; s < std::min(node->containedSplats->size(), sizeof(storageBlock[currentMemoryPosition].splatIds) / sizeof(uint32_t)); s++){
    //             if(node->containedSplats->size() > sizeof(storageBlock[currentMemoryPosition].splatIds) / sizeof(uint32_t)){
    //                 std::cout<<node->containedSplats->size()<<" "<<sizeof(storageBlock[currentMemoryPosition].splatIds) / sizeof(uint32_t)<<std::endl;
    //             }
    //             storageBlock[currentMemoryPosition].splatIds[s] = (*(node->containedSplats))[s];
    //         }

    //         storageBlock[currentMemoryPosition].representative = node->representative;
    //         storageBlock[currentMemoryPosition].flags = node->isLeaf;
    //         storageBlock[currentMemoryPosition].level = node->level;

    //         glm::vec3 center = (node->coverage[0] + node->coverage[1]) / 2.0f;

    //         storageBlock[currentMemoryPosition].center.x = center.x;
    //         storageBlock[currentMemoryPosition].center.y = center.y;
    //         storageBlock[currentMemoryPosition].center.z = center.z; 

    //         storageBlock[currentMemoryPosition].diagonal = glm::length((node->coverage[0] - node->coverage[1]));

    //         currentMemoryPosition++;

    //         for(HybridVH * child : node->children){
    //             process_queue.push_back(child);
    //         }

    //     }
	// }

    // printf("Built %d subtrees\n", roots.size());

    printf("Done building space partitioning\n");

    num_elements = sd.size();
    renderMask = (bool *)malloc(sizeof(bool) * num_elements);
    memset(renderMask, 0, sizeof(bool) * num_elements);

    end = std::chrono::steady_clock::now();
    int octreeTime = std::chrono::duration_cast<std::chrono::milliseconds>(end - begin).count();
    printf("Octree built in %f s\n", octreeTime / 1000.0f);
    
    printf("Number of splats: %d\n", num_elements);

    // checkCudaErrors(hipMalloc(&cudaStorageBlock, sizeof(CUDATreeNode) * totalStorageSize));
    // assert(cudaStorageBlock != NULL);
    // checkCudaErrors(hipMemcpy((void *)cudaStorageBlock, (void *)storageBlock, sizeof(CUDATreeNode) * totalStorageSize, hipMemcpyHostToDevice));

    // checkCudaErrors(hipMalloc(&cuda_roots, sizeof(uint32_t) * roots.size()));
    // assert(cuda_roots != NULL);
    // checkCudaErrors(hipMemcpy((void *)cuda_roots, (void *)roots.data(), sizeof(uint32_t) * roots.size(), hipMemcpyHostToDevice));

    /* Allocate and send splat data to GPU memory */
    SplatData * d_sd;
    checkCudaErrors(hipMalloc(&d_sd, sizeof(SplatData) * num_elements));
    assert(d_sd != NULL);
    checkCudaErrors(hipMemcpy((void*)d_sd, (void*) sd.data(), sizeof(SplatData) * num_elements, hipMemcpyHostToDevice));

    /* Allocate additional data buffers */
    float4 * d_conic_opacity;
    float3 * d_rgb;
    float2 * d_image_point;
    int * d_radius;
    float * d_depth;
    int * d_overlap;
    int * d_overlap_sums;
    bool * d_renderMask;
    float * d_cov3ds;

    checkCudaErrors(hipMalloc(&d_conic_opacity, sizeof(float4) * num_elements));
    assert(d_conic_opacity != NULL);

    checkCudaErrors(hipMalloc(&d_rgb, sizeof(float3) * num_elements));
    assert(d_rgb != NULL);

    checkCudaErrors(hipMalloc(&d_image_point, sizeof(float2) * num_elements));
    assert(d_image_point != NULL);

    checkCudaErrors(hipMalloc(&d_radius, sizeof(int) * num_elements));
    assert(d_radius != NULL);

    checkCudaErrors(hipMalloc(&d_depth, sizeof(float) * num_elements));
    assert(d_depth != NULL);

    checkCudaErrors(hipMalloc(&d_overlap, sizeof(int) * num_elements));
    assert(d_overlap != NULL);

    checkCudaErrors(hipMalloc(&d_overlap_sums, sizeof(int) * num_elements));
    assert(d_overlap_sums != NULL);

    checkCudaErrors(hipMalloc(&d_cov3ds, sizeof(int) * num_elements * 6));
    assert(d_cov3ds != NULL);

    checkCudaErrors(hipMalloc(&d_renderMask, sizeof(bool) * num_elements));
    assert(d_renderMask != NULL);
    checkCudaErrors(hipMemcpy(d_renderMask, renderMask, sizeof(bool) * num_elements, hipMemcpyHostToDevice));

    thrust::device_ptr<bool> d_renderMask_devPtr(d_renderMask);

    dim3 block(BLOCK_X, BLOCK_Y, 1); // One thread per pixel!
    dim3 grid(SCREEN_WIDTH / BLOCK_X + 1, SCREEN_HEIGHT / BLOCK_Y + 1, 1);

    uint32_t * d_tile_range_min;
    uint32_t * d_tile_range_max;

    checkCudaErrors(hipMalloc(&d_tile_range_min, sizeof(uint32_t) * grid.x * grid.y));
    checkCudaErrors(hipMalloc(&d_tile_range_max, sizeof(uint32_t) * grid.x * grid.y));

    uint64_t * d_sort_keys_in;
    uint64_t * d_sort_keys_out;
    uint32_t * d_sort_ids_in;
    uint32_t * d_sort_ids_out;
    checkCudaErrors(hipMalloc(&d_sort_keys_in, sizeof(uint64_t) * maxDuplicatedGaussians));
    checkCudaErrors(hipMalloc(&d_sort_keys_out, sizeof(uint64_t) * maxDuplicatedGaussians));
    checkCudaErrors(hipMalloc(&d_sort_ids_in, sizeof(uint32_t) * maxDuplicatedGaussians));
    checkCudaErrors(hipMalloc(&d_sort_ids_out, sizeof(uint32_t) * maxDuplicatedGaussians));

    /* Set up resources for texture writing */
    GLuint pboId;
    GLuint texId;
    GLfloat * imageData = new GLfloat[SCREEN_HEIGHT * SCREEN_WIDTH * 4];

    struct hipGraphicsResource * cuda_pbo_resource;
    void * d_pbo_buffer = NULL;

    // Initialize the texture
    glGenTextures(1, &texId);
    glBindTexture(GL_TEXTURE_2D, texId);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP);
    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA16F, SCREEN_WIDTH, SCREEN_HEIGHT, 0, GL_RGBA, GL_FLOAT, (GLvoid*)imageData);
    glBindTexture(GL_TEXTURE_2D, 0);

    // Initialize PBO
    glGenBuffers(1, &pboId);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pboId);
    glBufferData(GL_PIXEL_UNPACK_BUFFER, SCREEN_HEIGHT * SCREEN_WIDTH * 4 * sizeof(float), 0, GL_STREAM_DRAW);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);

    // Prepare CUDA interop
    checkCudaErrors(hipMalloc(&d_pbo_buffer, 4 * SCREEN_HEIGHT * SCREEN_WIDTH * sizeof(float)));
    checkCudaErrors(hipGraphicsGLRegisterBuffer(&cuda_pbo_resource, pboId, hipGraphicsRegisterFlagsNone));

    void     *d_temp_storage = NULL;
    size_t   temp_storage_bytes = 200000000;

    checkCudaErrors(hipMalloc(&d_temp_storage, temp_storage_bytes));
    

    /* Very basic FPS metrics */
    int currentFPSIndex = 0;

    getCameraParameters(0, cameraPosition, cameraRotation);

    begin = std::chrono::steady_clock::now();

    hipEvent_t kernelStart;
    hipEvent_t kernelEnd;

    checkCudaErrors(hipEventCreate(&kernelStart));
    checkCudaErrors(hipEventCreate(&kernelEnd));

    /* Main program loop */
    while (!glfwWindowShouldClose(window))
    {
        /* Clear color and depth buffers */
        glClear(GL_DEPTH_BUFFER_BIT | GL_COLOR_BUFFER_BIT);


        /* Bind the texture and Pixel Buffer */
        glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pboId);
        glBindTexture(GL_TEXTURE_2D, texId);
        
        glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0, SCREEN_WIDTH, SCREEN_HEIGHT, GL_RGBA, GL_FLOAT, 0);

        /* Map the OpenGL resources to a CUDA memory location */
        checkCudaErrors(hipGraphicsMapResources(1, &cuda_pbo_resource, 0));
        float4* dataPointer = nullptr;
        size_t num_bytes;
        checkCudaErrors(hipGraphicsResourceGetMappedPointer((void**)&dataPointer, &num_bytes, cuda_pbo_resource));
        assert(num_bytes >= SCREEN_HEIGHT * SCREEN_WIDTH * 4 * sizeof(float));
        assert(dataPointer != nullptr);

        auto softwareRasterizer = [&] (int forcedCameraIndex){
            /* --------- RENDERING ------------*/
            if(cameraMode == 1){
                getCameraParameters(forcedCameraIndex, cameraPosition, cameraRotation);
            }

            modelview = glm::lookAt(cameraPosition, cameraRotation * glm::vec3(0.0f, 0.0f, 1.0f) + cameraPosition, cameraRotation * glm::vec3(0.0f, 1.0f, 0.0f));
            perspective = glm::perspective(fovy, (float)SCREEN_WIDTH / (float)SCREEN_HEIGHT, 0.009f, 100.0f) * modelview;

            Frustum f;
            computeFrustum(f, perspective);

            int renderMode = (selectedViewMode<<4) + renderPrimitive;

            checkCudaErrors(hipMemset(d_renderMask, 0, sizeof(bool) * num_elements));
            // checkCudaErrors(hipMemset(d_renderMask, 1, sizeof(bool) * orig_num_splats / 4));
            memset(renderMask, 0, sizeof(bool) * num_elements);

            if(renderConfig.structure == std::string("octree")) 
                markForRender<GaussianOctree>(renderMask, static_cast<GaussianOctree*>(spacePartitioningRoot), autoLevel ? -1 : renderLevel, cameraPosition, fovy, SCREEN_WIDTH, diagonalProjectionThreshold, num_elements);
            else if(renderConfig.structure == std::string("bvh"))
                markForRender<GaussianBVH>(renderMask, static_cast<GaussianBVH*>(spacePartitioningRoot), autoLevel ? -1 : renderLevel, cameraPosition, fovy, SCREEN_WIDTH, diagonalProjectionThreshold, num_elements);
            else
                markForRender<HybridVH>(renderMask, static_cast<HybridVH*>(spacePartitioningRoot), autoLevel ? -1 : renderLevel, cameraPosition, fovy, SCREEN_WIDTH, diagonalProjectionThreshold, num_elements);

            // markForRender(renderMask, nodes, autoLevel ? -1 : renderLevel, cameraPosition, fovy, SCREEN_WIDTH, diagonalProjectionThreshold, num_elements);
            hipMemcpy(d_renderMask, renderMask, sizeof(bool) * num_elements, hipMemcpyHostToDevice);

            checkCudaErrors(hipEventRecord(kernelStart));
            // CUDAmarkForRender<<<roots.size() / 256 + 1, 256>>>(d_renderMask, cudaStorageBlock, cuda_roots, roots.size(), cameraPosition, fovy, SCREEN_WIDTH, diagonalProjectionThreshold, f, useFrustumCulling, (!autoLevel)*renderLevel);
            checkCudaErrors(hipDeviceSynchronize());
            checkCudaErrors(hipEventRecord(kernelEnd));
            checkCudaErrors(hipEventSynchronize(kernelEnd));
            checkCudaErrors(hipEventElapsedTime(&traversalTime, kernelStart, kernelEnd));
            avgTraversal.insert(traversalTime);

            checkCudaErrors(hipEventRecord(kernelStart));

            preprocessGaussians<<<num_elements / LINE_BLOCK + 1, LINE_BLOCK>>>(num_elements, d_sd, perspective, modelview, cameraPosition, fovy, fovx, d_conic_opacity, d_rgb, d_image_point, d_radius, d_depth, d_overlap, SCREEN_WIDTH, SCREEN_HEIGHT, grid, renderMode, d_renderMask);
            checkCudaErrors(hipDeviceSynchronize());

            renderedSplats = thrust::count(d_renderMask_devPtr, d_renderMask_devPtr + num_elements, true);
            
            // Run inclusive prefix sum
            hipcub::DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes, d_overlap, d_overlap_sums, num_elements);

            int totalDuplicateGaussians = 0;
            checkCudaErrors(hipMemcpy(&totalDuplicateGaussians, d_overlap_sums + num_elements - 1, sizeof(int), hipMemcpyDeviceToHost));

            totalDuplicateGaussians = min(totalDuplicateGaussians, maxDuplicatedGaussians);

            /* Populate sorting keys array */
            duplicateGaussians<<<num_elements / LINE_BLOCK + 1, LINE_BLOCK>>>(num_elements, d_image_point, d_radius, d_depth, d_overlap_sums, d_sort_keys_in, d_sort_ids_in, grid);
            checkCudaErrors(hipDeviceSynchronize());

            /* TODO: determine highest MSB to pass to sorting, so we don't use all 64 bits */
            hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_sort_keys_in, d_sort_keys_out, d_sort_ids_in, d_sort_ids_out, totalDuplicateGaussians);

            checkCudaErrors(hipMemset(d_tile_range_min, 0, sizeof(uint32_t) * grid.x * grid.y));
            checkCudaErrors(hipMemset(d_tile_range_max, 0, sizeof(uint32_t) * grid.x * grid.y));

            getTileRanges<<<(totalDuplicateGaussians) / LINE_BLOCK + 1, LINE_BLOCK>>>(d_sort_keys_out, totalDuplicateGaussians, d_tile_range_min, d_tile_range_max);
            checkCudaErrors(hipDeviceSynchronize());

            checkCudaErrors(hipEventRecord(kernelEnd));
            checkCudaErrors(hipEventSynchronize(kernelEnd));
            checkCudaErrors(hipEventElapsedTime(&prepTime, kernelStart, kernelEnd));
            avgPrep.insert(prepTime);

            checkCudaErrors(hipEventRecord(kernelStart));
            render<<<grid, block>>>(num_elements, d_sd, d_conic_opacity, d_rgb, d_image_point, d_depth, d_tile_range_min, d_tile_range_max, d_sort_ids_out, SCREEN_WIDTH, SCREEN_HEIGHT, grid, dataPointer);
            checkCudaErrors(hipDeviceSynchronize());
            checkCudaErrors(hipEventRecord(kernelEnd));
            checkCudaErrors(hipEventSynchronize(kernelEnd));
            checkCudaErrors(hipEventElapsedTime(&renderTime, kernelStart, kernelEnd));
            avgRender.insert(renderTime);
        };

        auto saveRenderRoutine = [&](const char * filename){
            std::vector<float> floatPixelData(SCREEN_HEIGHT * SCREEN_WIDTH * 4);
            std::vector<unsigned char> pixelData(SCREEN_HEIGHT * SCREEN_WIDTH * 4);

            hipMemcpy(floatPixelData.data(), dataPointer, SCREEN_HEIGHT * SCREEN_WIDTH * 4 * sizeof(float), hipMemcpyDeviceToHost);

            for (size_t i = 0; i < pixelData.size(); ++i) {
                pixelData[i] = static_cast<unsigned char>(std::min(1.0f, floatPixelData[i]) * 255.0f);
            }

            // Write image to file
            stbi_flip_vertically_on_write(true);                
            stbi_write_png(filename, SCREEN_WIDTH, SCREEN_HEIGHT, 4, pixelData.data(), SCREEN_WIDTH * 4);
        };

        if(recordExperiment){
            cameraMode = 1;
            std::vector<float> traversalTimeVector;
            std::vector<float> prepTimeVector;
            std::vector<float> renderTimeVector;
            std::vector<uint32_t> numRenSplatsVector;
            char filename[64];

            for(int i = 0; i < cameraData.size(); i++){
                /* Color */
                selectedViewMode = 0;
                softwareRasterizer(i);
                traversalTimeVector.push_back(traversalTime);
                prepTimeVector.push_back(prepTime);
                renderTimeVector.push_back(renderTime);
                numRenSplatsVector.push_back(renderedSplats);
                snprintf(filename, 64, "renders/%05d.png", i);
                saveRenderRoutine(filename);
                /* Depth */
                selectedViewMode = 1;
                softwareRasterizer(i);
                snprintf(filename, 64, "renders/d%05d.png", i);
                saveRenderRoutine(filename);
            }
            snprintf(filename, 64, "renders/stats_%.2f_%d.txt", diagonalProjectionThreshold, useFrustumCulling);
            std::ofstream csv_out(filename);
            if(csv_out.is_open()){
                csv_out << "FrameIdx,RenderedSplats,TraversalTime,PrepTime,RenderTime" << std::endl;
                for(int i = 0; i < cameraData.size(); i++){
                    csv_out << i << "," << numRenSplatsVector[i] 
                                 << "," << traversalTimeVector[i]
                                 << "," << prepTimeVector[i]
                                 << "," << renderTimeVector[i]
                                 << std::endl;
                }
                csv_out.close();
            }
        }

        if(batchRender){
            cameraMode = 1;
            for(int i = 0; i < cameraData.size(); i+=10){
                char filename[64];
                /* Color */
                selectedViewMode = 0;
                softwareRasterizer(i);
                snprintf(filename, 64, "renders/%05d.png", i);
                saveRenderRoutine(filename);
                /* Depth */
                selectedViewMode = 1;
                softwareRasterizer(i);
                snprintf(filename, 64, "renders/d%05d.png", i);
                saveRenderRoutine(filename);
            }
        }
        else{
            softwareRasterizer(cameraIndex);
        }

        /* Build ImGui interface */
        buildInterface();

        if(saveRender){
            auto t = std::time(nullptr);
            auto tm = *std::localtime(&t);

            std::ostringstream oss;
            oss << "renders/output_" << std::put_time(&tm, "%d-%m %H-%M-%S")<<".png";
            auto str = oss.str();
            saveRenderRoutine(str.c_str());
        }

        // diagonalProjectionThreshold = 300.0f;

        /* Unmap the OpenGL resources */
        checkCudaErrors(hipGraphicsUnmapResources(1, &cuda_pbo_resource, 0));

        /* Draw a quad which covers the entire screen */

        glBindTexture(GL_TEXTURE_2D, texId);
        glEnable(GL_TEXTURE_2D);

        glBegin(GL_QUADS);
        glNormal3f(0, 0, 1);
        glTexCoord2f(0.0f, 0.0f);   glVertex3f(-1.0f, -1.0f, 0.0f);
        glTexCoord2f(1.0f, 0.0f);   glVertex3f( 1.0f, -1.0f, 0.0f);
        glTexCoord2f(1.0f, 1.0f);   glVertex3f( 1.0f,  1.0f, 0.0f);
        glTexCoord2f(0.0f, 1.0f);   glVertex3f(-1.0f,  1.0f, 0.0f);
        glEnd();

        /* Unbind the texture and PBO */
        glBindTexture(GL_TEXTURE_2D, 0);
        glDisable(GL_TEXTURE_2D);

        renderInterface();

        /* Swap buffers and handle GLFW events */
        glfwSwapBuffers(window);
        glfwPollEvents();

        /* Compute and display FPS every set number of frames */
        currentFPSIndex++;
        if(currentFPSIndex == FPS_COUNTER_REFRESH){
            end = std::chrono::steady_clock::now();
            int milisCount = std::chrono::duration_cast<std::chrono::milliseconds>(end - begin).count();
            char title[256];
            sprintf(title, "AccelerateGS [FPS: %f]", 1000.0f / milisCount * FPS_COUNTER_REFRESH);
            glfwSetWindowTitle(window, title);
            begin = std::chrono::steady_clock::now();
            currentFPSIndex = 0;
        }
    }

    shutdownIMGui();

    /* Unmap resources and free allocated memory */
    checkCudaErrors(hipGraphicsUnregisterResource(cuda_pbo_resource));
    glDeleteTextures(1, &texId);
    glDeleteBuffers(1, &pboId);
    hipFree(d_sd);
    hipFree(d_pbo_buffer);

    hipFree(d_conic_opacity);
    hipFree(d_rgb);
    hipFree(d_image_point);
    hipFree(d_radius);
    hipFree(d_depth);
    hipFree(d_overlap);
    hipFree(d_overlap_sums);
    hipFree(d_cov3ds);
    hipFree(d_renderMask);

    checkCudaErrors(hipFree(d_tile_range_min));
    checkCudaErrors(hipFree(d_tile_range_max));

    checkCudaErrors(hipFree(d_sort_keys_in));
    checkCudaErrors(hipFree(d_sort_keys_out));
    checkCudaErrors(hipFree(d_sort_ids_in));
    checkCudaErrors(hipFree(d_sort_ids_out));
    checkCudaErrors(hipFree(d_temp_storage));

    if(spacePartitioningRoot != nullptr) delete spacePartitioningRoot;
    delete [] imageData;
    free(renderMask);

    glfwDestroyWindow(window);
    glfwTerminate();

    return 0;
}